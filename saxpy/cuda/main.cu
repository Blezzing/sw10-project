
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

typedef std::chrono::high_resolution_clock Clock;

__global__ void kernel(int n, float a, float* x, float* y){
    for( int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){
        x[i] = a * x[i] + y[i];
    }
}

int main(void){
    int N = 1 << 29;
    float a = 11.0;

    float *h_x;
    float *h_y; 
    float *d_x;
    float *d_y;
    h_x = (float*)malloc(N*sizeof(float));
    h_y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    for(int i = 0; i < N; i++){
        h_x[i] = rand();
        h_y[i] = rand();
    }

    hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N*sizeof(float), hipMemcpyHostToDevice);


    auto t0 = Clock::now();
    kernel<<<128, 128>>>(N, a, d_x, d_y);
    auto t1 = Clock::now();

    hipMemcpy(h_x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    std::cout
            << "elapsed: " << std::chrono::duration_cast<std::chrono::nanoseconds>(t1 - t0).count() << "ns" << std::endl;
       
}
